#include "common_utils/IncludeImplCUDA.hpp"

// This file is only meant to compile the necessary CUDA methods.

// To enable Google Tests, we compile all tests with cpp compiler, and
// separately compile only CUDA code here. Otherwise, this precompilation
// would not be necessary.
#include "common_utils/memory_managment/MemoryWrapperCUDA.cuh"
#include "common_utils/memory_managment/MemoryContainerCUDA.cuh"
template class common_utils::MemoryWrapper<double>;
template class common_utils::MemoryWrapper<float>;
template class common_utils::MemoryContainer<double>;
template class common_utils::MemoryContainer<float>;